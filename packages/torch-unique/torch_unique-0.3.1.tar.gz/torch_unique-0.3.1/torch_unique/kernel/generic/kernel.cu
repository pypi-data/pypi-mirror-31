
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/kernel.cu"
#else

void unique_()(THCState *state, THCTensor *input) {
  input = THCTensor_(newContiguous)(state, input);

  thrust::device_ptr<real> first(THCTensor_(data)(state, input));
  ptrdiff_t numel = THCTensor_(nElement)(state, input);

  THRUST_ALLOC(state);
  THRUST_EXEC(thrust::sort, first, first + numel, ThrustLTOp<real>());
  thrust::device_ptr<real> last(THRUST_EXEC(thrust::unique, first, first + numel, ThrustEQOp<real>()));

  numel = last - first;
  THCTensor_(resize1d)(state, input, numel);

  THCTensor_(free)(state, input);
}

void unique_(ByKey)(THCState *state, THCTensor *key, THCTensor *value) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, key, value));

  key = THCTensor_(newContiguous)(state, key);
  value = THCTensor_(newContiguous)(state, value);

  thrust::device_ptr<real> firstKey(THCTensor_(data)(state, key));
  thrust::device_ptr<real> firstValue(THCTensor_(data)(state, value));
  ptrdiff_t numel = THCTensor_(nElement)(state, key);

  THRUST_ALLOC(state);
  THRUST_EXEC(thrust::sort_by_key, firstKey, firstKey + numel, firstValue, ThrustLTOp<real>());
  thrust::pair<thrust::device_ptr<real>, thrust::device_ptr<real> > last(THRUST_EXEC(thrust::unique_by_key, firstKey, firstKey + numel, firstValue, ThrustEQOp<real>()));

  THCTensor_(resize1d)(state, key, last.first - firstKey);
  THCTensor_(resize1d)(state, value, last.second - firstValue);

  THCTensor_(free)(state, key);
  THCTensor_(free)(state, value);
}

#endif
